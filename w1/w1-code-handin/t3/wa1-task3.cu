#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>
// Cuda testing and assertion from Anders
#include <assert.h>
#define cudaAssert(x) (assert((x) == hipSuccess))

// Custom and default block_size and N
#ifndef BLOCK_SIZE
  #define BLOCK_SIZE 256
#endif
#ifndef N_ELEMS
  #define N_ELEMS 753411
#endif
// How many benchmarks to run
#define BENCH_RUNS 200


// Src: Lab1-CudaIntro. Get time difference
int timeval_subtract( 
        struct timeval *result,
        struct timeval *t2,
        struct timeval *t1)
{
  unsigned int resolution = 1000000;
  long int diff = (t2->tv_usec + resolution * t2->tv_sec) - 
                  (t1->tv_usec + resolution * t2->tv_sec);
  result->tv_sec = diff / resolution; result->tv_usec = diff % resolution;
  return (diff<0);
}

__global__ void kernel(float *d_in, float *d_out, int N){
  const unsigned int lid = threadIdx.x; // Local id inside a block
  const unsigned int gid = blockIdx.x*blockDim.x + lid; // global id
  if (gid < N){
    float x = d_in[gid]/(d_in[gid]-2.3);
    d_out[gid] = x*x*x;
  }
}

int gpu_run(float* inp, float* out, int N)
{
  // Most of this code is stolened from the lab1 slides
  // Time tracking vars
  unsigned long int elapsed; 
  struct timeval t_start, t_end, t_diff;

  // Block distr vars
  unsigned int block_size = BLOCK_SIZE;
  unsigned int num_blocks = ((N + (block_size - 1)) / block_size);

  // Memory assignment
  unsigned int mem_size = N*sizeof(float);
  float* d_in;
  float* d_out;
  hipMalloc((void**)&d_in, mem_size);
  hipMalloc((void**)&d_out, mem_size);

  // Copy host mem to device
  hipError_t e = hipMemcpy(d_in, inp, mem_size, hipMemcpyHostToDevice);
  if ( e != 0)
  {
    printf("Cuda memory couldn't be allocated. Error:\n%s\n", hipGetErrorString(e));
    return 1;
  }
  // Exec kernel(with timetrack)
  gettimeofday(&t_start, NULL);
  for(int i = 0; i < BENCH_RUNS; i++){
    kernel<<<num_blocks, block_size>>>(d_in, d_out, N);
  }
  cudaAssert(hipPeekAtLastError());
  hipDeviceSynchronize();// Ensure kernel has finished
  gettimeofday(&t_end, NULL);
  // Copy result from device to host
  hipMemcpy(out, d_out, mem_size, hipMemcpyDeviceToHost);
  hipFree(d_in); hipFree(d_out);
  // Calculate and print time
  timeval_subtract(&t_diff, &t_end, &t_start);
  elapsed = ((t_diff.tv_sec*1e6+t_diff.tv_usec) / BENCH_RUNS);
  printf("GPU(%d runs) took %d microseconds (%.2fms)\n",
          BENCH_RUNS,
          elapsed,
          elapsed / 1000.0
        );
  return 0;
}

void seq_run(float* inp, float* out, int N){
  unsigned long int elapsed; 
  struct timeval t_start, t_end, t_diff;
  gettimeofday(&t_start, NULL);
  for(unsigned int j = 0; j < BENCH_RUNS; j++){
    for(unsigned int i = 0; i < N; ++i){
      float x = inp[i]/(inp[i]-2.3);
      out[i] = x*x*x;
    }
  }
  gettimeofday(&t_end, NULL);
  timeval_subtract(&t_diff, &t_end, &t_start);
  elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / BENCH_RUNS;
  printf("CPU(%d runs) took %d microseconds (%.2fms)\n",
          BENCH_RUNS,
          elapsed,
          elapsed / 1000.0
        );
}

int main( int argc, char** argv){
  unsigned int N = N_ELEMS;
  unsigned int mem_size = N*sizeof(float);
  // Init memory arrays
  float* in = (float*) malloc(mem_size);
  float* gpu_out = (float*) malloc(mem_size);
  float* seq_out = (float*) malloc(mem_size);
  // And init the input array
  for (unsigned int i=0; i<N; ++i) in[i] = (float)i;

  // Run the code on the CPU
  seq_run(in, seq_out, N);
  // Run the code on the GPU
  int e = gpu_run(in, gpu_out, N);
  if (e != 0){
    printf("Error in gpu run\n");
    return 1;
  }

  // Now validate results:
  int passed = 0;
  int invalid = 0;
  for (int i = 0; i < N; ++i) {
    if (fabs(seq_out[i] - gpu_out[i]) < 0.0001)
        passed++;
    else invalid++;
  }
  printf("Passed: %06d, Invalid: %06d\n", passed, invalid);

  //DEBUG: Print the first 10 and last 10 values to 10p of precision
  // for(int i = 0; i < 10; i++) printf("%6d:\t%.10f\t%.10f\n", i, seq_out[i], gpu_out[i]);
  // for(int i = 0; i < 10; i++) printf("%6d:\t%.10f\t%.10f\n", N-i, seq_out[N-i], gpu_out[N-i]);
  // Free outpus databases
  free(in); free(gpu_out); free(seq_out);

  return 0;
}
